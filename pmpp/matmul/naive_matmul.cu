#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// Figure 3.11 from PMPP
// A matrix muliplication kernel using one thread to compute one P element.
__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < Width) && (col < Width)) {
        float Pvalue = 0;
        for (int k = 0; k < Width; ++k) {
            Pvalue += M[row*Width+k] * N[k*Width+col];
        }
        P[row*Width+col] = Pvalue;
    }
}

// PMPP Exercise 1a
__global__ void matmul_exercise_1a(float *M, float *N, float *P, int Width) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < Width) {
        for (int p = 0; p < Width; p++) {
            P[row * P + p] = 0.0f;
        }

        for (int n = 0; n < Width; n++) {
            for (int p = 0; p < Width; p++) {
                P[row * Width + p] += M[row * Width + n] * N[n * Width + p];
            }
        }
    }
}

// allocate memory for matmul and manage data transfer between host and device
void matmul(float* Min, float* Nin, float* Pout, int width) {
  float *Min_d, *Nin_d, *Pout_d;
  int size = width * width * sizeof(float);

  hipMalloc((void**) &Min_d, size);
  hipMalloc((void**) &Nin_d, size);
  hipMalloc((void**) &Pout_d, size);

  hipMemcpy(Min_d, Min, size, hipMemcpyHostToDevice);
  hipMemcpy(Nin_d, Nin, size, hipMemcpyHostToDevice);
  hipMemcpy(Pout_d, Pout, size, hipMemcpyHostToDevice);

  // Dynamically allocate threads dependant on image size
  dim3 dimGrid(ceil(width*width / 16.0), ceil(width*width / 16.0), 1);
  dim3 dimBlock(16, 16, 1);

  MatrixMulKernel<<<dimGrid, dimBlock>>>(Min_d, Nin_d, Pout_d, width);
  
  hipMemcpy(Pout, Pout_d, size, hipMemcpyDeviceToHost);

  hipFree(Min_d);
  hipFree(Nin_d);
  hipFree(Pout_d);
}

// print the given array
void print_array(float* arr, int size) {
  for (int i = 0; i < size; ++i) {
    if (i > 0) {
      printf(", ");
      if (i % 10 == 0) {
        printf("\n");
      }
    }
    printf("%0.1f", arr[i]);
  }
  printf("\n"); 
}

int main(void) {
  const int n = 9;
  float A[n];
  float B[n];
  float C[n];

  // set values of input arrays
  for (int i = 0; i < n; ++i) {
    A[i] = i;
    B[i] = i;
  }

  print_array(A, n);
  print_array(B, n);

  matmul(A, B, C, 3);

  print_array(C, n);

  return 0;
}
