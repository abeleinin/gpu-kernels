#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "../utils.h"

__global__ void guardKernel(float* a, float* out, int n) {
  int i = threadIdx.x;
  if (i < n) {
    out[i] = a[i] + 10;
  }
}

void guardTest(float* a, float* out, int n) {
  float *a_d, *out_d;
  int size = n * sizeof(float);

  hipMalloc((void**) &a_d, size);
  hipMalloc((void**) &out_d, size);

  hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);

  guardKernel<<<1, 8>>>(a_d, out_d, n);
  
  hipMemcpy(out, out_d, size, hipMemcpyDeviceToHost);

  hipFree(a_d);
  hipFree(out_d);
}


int main(void) {
  const int n = 4;
  float a[n];
  float out[n];

  init_arr(a, n);

  guardTest(a, out, n);

  print_arr(out, n);

  return 0;
}

