#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "../utils.h"

__global__ void guard_kernel(float* a, float* out, int n) {
  int i = threadIdx.x;
  if (i < n) {
    out[i] = a[i] + 10;
  }
}

void guard_test(float* a, float* out, int n) {
  float *a_d, *out_d;
  int size = n * sizeof(float);

  hipMalloc((void**) &a_d, size);
  hipMalloc((void**) &out_d, size);

  hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);

  guard_kernel<<<1, 8>>>(a_d, out_d, n);
  
  hipMemcpy(out, out_d, size, hipMemcpyDeviceToHost);

  hipFree(a_d);
  hipFree(out_d);
}


int main(void) {
  int n = 4;
  float a[n];
  float out[n];

  arange_array(a, n);

  guard_test(a, out, n);

  print_array(out, n);

  return 0;
}

