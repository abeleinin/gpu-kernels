#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "utils.h"

__global__ void zip_kernel(float* a, float* b, float* out, int n) {
  int i = threadIdx.x;
  out[i] = a[i] + b[i];
}

void zip_test(float* a, float* b, float* out, int n) {
  float *a_d, *b_d, *out_d;
  int size = n * sizeof(float);

  hipMalloc((void**) &a_d, size);
  hipMalloc((void**) &b_d, size);
  hipMalloc((void**) &out_d, size);

  hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b, size, hipMemcpyHostToDevice);

  zip_kernel<<<1, 4>>>(a_d, b_d, out_d, n);
  
  hipMemcpy(out, out_d, size, hipMemcpyDeviceToHost);

  hipFree(a_d);
  hipFree(b_d);
  hipFree(out_d);
}


int main(void) {
  int n = 4;
  float a[n];
  float b[n];
  float out[n];

  arange_array(a, n);
  arange_array(b, n);

  zip_test(a, b, out, n);

  print_array(out, n);

  return 0;
}

