#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "utils.h"

__global__ void map_kernel(float* a, float* out, int n) {
  int i = threadIdx.x;
  out[i] = a[i] + 10;
}

void map_test(float* a, float* out, int n) {
  float *a_d, *out_d;
  int size = n * sizeof(float);

  hipMalloc((void**) &a_d, size);
  hipMalloc((void**) &out_d, size);

  hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);

  map_kernel<<<1, 4>>>(a_d, out_d, n);
  
  hipMemcpy(out, out_d, size, hipMemcpyDeviceToHost);

  hipFree(a_d);
  hipFree(out_d);
}


int main(void) {
  const int n = 4;
  float a[n];
  float out[n];

  arange_array(a, n);

  map_test(a, out, n);

  print_array(out, n);

  return 0;
}

