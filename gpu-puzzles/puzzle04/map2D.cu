#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "../utils.h"

__global__ void map_2D_kernel(float* a, float* out, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < n && j < n) {
    int index = j*n + i;
    out[index] = a[index] + 10;
  }
}

void map_2D_test(float* a, float* out, int n) {
  float *a_d, *out_d;
  int size = n * n * sizeof(float);

  hipMalloc((void**) &a_d, size);
  hipMalloc((void**) &out_d, size);

  hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);

  dim3 dimGrid(3, 3, 1);
  dim3 dimBlock(1, 1, 1);
  map_2D_kernel<<<dimGrid, dimBlock>>>(a_d, out_d, n);
  
  hipMemcpy(out, out_d, size, hipMemcpyDeviceToHost);

  hipFree(a_d);
  hipFree(out_d);
}


int main(void) {
  const int N = 2;
  const int SIZE = 4;
  
  float a[N][N];
  float out[N][N];

  arange_array((float*)a, SIZE);

  map_2D_test((float*)a, (float*)out, N);

  print_array((float*)out, SIZE);

  return 0;
}

