#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "../utils.h"

__global__ void mapKernel(float* a, float* out, int n) {
  int i = threadIdx.x;
  out[i] = a[i] + 10;
}

void mapTest(float* a, float* out, int n) {
  float *a_d, *out_d;
  int size = n * sizeof(float);

  hipMalloc((void**) &a_d, size);
  hipMalloc((void**) &out_d, size);

  hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);

  mapKernel<<<1, 4>>>(a_d, out_d, n);
  
  hipMemcpy(out, out_d, size, hipMemcpyDeviceToHost);

  hipFree(a_d);
  hipFree(out_d);
}


int main(void) {
  const int n = 4;
  float a[n];
  float out[n];

  init_arr(a, n);

  mapTest(a, out, n);

  print_arr(out, n);

  return 0;
}

